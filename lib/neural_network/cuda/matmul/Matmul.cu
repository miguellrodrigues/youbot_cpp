#include "hip/hip_runtime.h"
//
// Created by miguel on 11/03/2021.
//

#include "Matmul.cuh"

#define BLOCK_SIZE 16

__global__ void matrixMultiplyKernel(const double *a, const double *b, double *c, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    double aux = .0;

    if (row < m && col < k) {
        for (int i = 0; i < n; i++) {
            aux += a[row * n + i] * b[i * k + col];
        }

        c[row * k + col] = aux;
    }
}

void Matmul::matrixMultiply(double *a, double *b, double *c, int m, int n, int k) {
    dim3 dimGrid((k + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

/*    if (n*n > 512) {
        threadsPerBlock.x = 512;
        threadsPerBlock.y = 512;
        blocksPerGrid.x = ceil(double(n)/double(threadsPerBlock.x));
        blocksPerGrid.y = ceil(double(n)/double(threadsPerBlock.y));
    }*/

    matrixMultiplyKernel<<<dimGrid, dimBlock>>>(a, b, c, m, n, k);
}
