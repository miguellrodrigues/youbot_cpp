#include "hip/hip_runtime.h"
//
// Created by miguel on 11/03/2021.
//

#include "MatUtil.cuh"
#include <stdio.h>

#define BLOCK_SIZE 16

__global__ void matrixMultiplyKernel(const double *a, const double *b, double *c, int m, int n, int k) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < k) {
        double aux = .0;

        for (int i = 0; i < n; i++) {
            aux += a[row * n + i] * b[i * k + col];
        }

        c[row * k + col] = aux;
    }
}

__global__ void matrixTransposeKernel(double *a, double *b, unsigned int rows, unsigned int cols) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < cols && idy < rows) {
        unsigned int pos = idy * cols + idx;
        unsigned int transposedPos = idx * rows + idy;

        b[transposedPos] = a[pos];
    }
}

void MatUtil::matrixMultiply(double *a, double *b, double *c, int m, int n, int k) {
    dim3 dimGrid((k + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    matrixMultiplyKernel<<<dimGrid, dimBlock>>>(a, b, c, m, n, k);
}

void MatUtil::matrixTranspose(double *a, double *b, unsigned int rows, unsigned int cols) {
    unsigned int n = rows * cols;

    dim3 dim_grid((n - 1) / BLOCK_SIZE + 1, (n - 1) / BLOCK_SIZE + 1, 1);
    dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);

    matrixTransposeKernel<<<dim_grid, dim_block>>>(a, b, rows, cols);
}
